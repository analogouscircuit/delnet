#include <stdio.h>
#include <hip/hip_runtime.h>


extern "C" __host__
void cuAlloc(void **block, size_t numelements, size_t typesize, int commRank)
{
	hipError_t cE;

	int cudaDeviceCount;

	// Check if enough devices
	if ( (cE = hipGetDeviceCount( &cudaDeviceCount )) != hipSuccess ) {
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
				cE, cudaDeviceCount);
		exit(-1);
	}

	// Assign rank to appropriate device
	if ( (cE = hipSetDevice( commRank % cudaDeviceCount )) != hipSuccess) {
		printf(" Unable to have rank %d set to cuda %d, error is %d \n",
				commRank, (commRank % cudaDeviceCount), cE);
		exit(-1);
	}

	cE = hipMallocManaged(block, numelements*typesize);
	if (cE != hipSuccess) {
		printf("%s\n", hipGetErrorName(cE));
		printf("%s\n", hipGetErrorString(cE));
	}
}

extern "C" __host__
void cuAllocDouble(double **block, size_t numelements, int commRank)
{
	size_t i;
	hipError_t cE;

	int cudaDeviceCount;

	// Check if enough devices
	if ( (cE = hipGetDeviceCount( &cudaDeviceCount )) != hipSuccess ) {
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
				cE, cudaDeviceCount);
		exit(-1);
	}

	// Assign rank to appropriate device
	if( (cE = hipSetDevice( commRank % cudaDeviceCount )) != hipSuccess) {
		printf(" Unable to have rank %d set to cuda %d, error is %d \n",
				commRank, (commRank % cudaDeviceCount), cE);
		exit(-1);
	}

	cE = hipMallocManaged(block, numelements*sizeof(double));
	if (cE != hipSuccess) {
		printf("%s\n", hipGetErrorName(cE));
		printf("%s\n", hipGetErrorString(cE));
	}

	/* initialize to 0 */
	for (i=0; i<numelements; i++) (*block)[i] = 0.0;

}

extern "C" __host__
void cuFreeDouble(double *block)
{
	hipError_t cE;
	
	cE = hipFree(block);
	if (cE != hipSuccess) {
		printf("%s\n", hipGetErrorName(cE));
		printf("%s\n", hipGetErrorString(cE));
	}
}

extern "C" __host__
void cuFree(void *block)
{
	hipError_t cE;

	cE = hipFree(block);
	if (cE != hipSuccess) {
		printf("%s\n", hipGetErrorName(cE));
		printf("%s\n", hipGetErrorString(cE));
	}
}
