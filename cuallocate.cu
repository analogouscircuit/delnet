#include <stdio.h>
#include <hip/hip_runtime.h>


extern "C" __host__
void cuAlloc(void **block, size_t numelements, size_t typesize, int commRank)
{
	hipError_t cE;

	int cudaDeviceCount;

	// Check if enough devices
	if ( (cE = hipGetDeviceCount( &cudaDeviceCount )) != hipSuccess ) {
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
				cE, cudaDeviceCount);
		exit(-1);
	}

	// Assign rank to appropriate device
	if ( (cE = hipSetDevice( commRank % cudaDeviceCount )) != hipSuccess) {
		printf(" Unable to have rank %d set to cuda %d, error is %d \n",
				commRank, (commRank % cudaDeviceCount), cE);
		exit(-1);
	}

	hipMallocManaged(block, numelements*typesize);
}

extern "C" __host__
void cuAllocDouble(double **block, size_t numelements, int commRank)
{
	size_t i;
	hipError_t cE;

	int cudaDeviceCount;

	// Check if enough devices
	if ( (cE = hipGetDeviceCount( &cudaDeviceCount )) != hipSuccess ) {
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
				cE, cudaDeviceCount);
		exit(-1);
	}

	// Assign rank to appropriate device
	if( (cE = hipSetDevice( commRank % cudaDeviceCount )) != hipSuccess) {
		printf(" Unable to have rank %d set to cuda %d, error is %d \n",
				commRank, (commRank % cudaDeviceCount), cE);
		exit(-1);
	}

	hipMallocManaged(block, numelements*sizeof(double));
	for (i=0; i<numelements; i++) (*block)[i] = 0.0;

}

extern "C" __host__
void cuFreeDouble(double *block)
{
	hipFree(block);
}

extern "C" __host__
void cuFree(void *block)
{
	hipFree(block);
}
