#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "delnetmpi.h"
#include "simutilsmpi.h"
//#include "simkernelsmpicuda.h"

#define TPB 512

/* -------------------- Random Sampling -------------------- */

/*
 * Samples from an exponential distribution. For generating 
 * Poissonian noise.
 */
extern "C"
double sk_mpi_expsampl(double lambda)
{
	return -log( (((double) rand()) / ((double) RAND_MAX + 1.0)))/lambda;
}



/* -------------------- Neuron Equations -------------------- */

/*
 * First function for Runge-Kutta method. This is the Izhikevich
 * "simple" model, voltage variable.
 */

extern "C"
static inline FLOAT_T f1(FLOAT_T v, FLOAT_T u, FLOAT_T input) {
	return (0.04*v + 5.0)*v + 140.0 - u + input;
}

/*
 * Second function for Runge-Kutta method. This is the Izhikevich
 * "simple" model, recover variable.
 */

extern "C"
static inline FLOAT_T f2(FLOAT_T v, FLOAT_T u, FLOAT_T a) {
	return a*(0.2*v - u);
}
/*
 * Update neuron state using 4th order Runge-Kutta
 */

extern "C"
void neuronupdate_rk4(FLOAT_T *v, FLOAT_T *u, FLOAT_T input, FLOAT_T a, FLOAT_T h) {
	FLOAT_T K1, K2, K3, K4, L1, L2, L3, L4, half_h, sixth_h;

	half_h = h*0.5;
	sixth_h = h/6.0;
	
	K1 = f1(*v, *u, 0.0);
	L1 = f2(*v, *u, a);

	K2 = f1(*v + half_h*K1, *u + half_h*L1, 0.0); 
	L2 = f2(*v + half_h*K1, *u + half_h*L1, a);

	K3 = f1(*v + half_h*K2, *u + half_h*L2, 0.0);
	L3 = f2(*v + half_h*K2, *u + half_h*L2, a);

	K4 = f1(*v + h*K3, *u + h*L3, 0.0);
	L4 = f2(*v + h*K3, *u + h*L3, a);

	*v = *v + sixth_h * (K1 + 2*K2 + 2*K3 + K4) + input;
	*u = *u + sixth_h * (L1 + 2*L2 + 2*L3 + L4); 
}

/*-------------------- Kernels -------------------- */

/*
 * Takes the neuron inputs, multiples them by appropriate
 * synaptic weight, sums, and returns result.
 */
extern "C"
void sk_mpi_getinputs(FLOAT_T *neuroninputs, dn_mpi_delaynet *dn, FLOAT_T *synapses)
{
	size_t k,j;
	FLOAT_T *delayoutputs;
	for (k=0; k<dn->num_nodes_l; k++) {
		// get inputs to neuron (outputs of delaylines)
		neuroninputs[k] = 0.0;
		delayoutputs = dn_mpi_getinputaddress(k,dn); //dn->outputs
		for (j=0; j < dn->nodes[k].num_in; j++) {
			neuroninputs[k] += delayoutputs[j] * synapses[ dn->nodes[k].idx_outbuf+j ];
		}
	}
}


/*
 * An update function for generating Poissonian input noise.
 */
extern "C"
unsigned int sk_mpi_poisnoise(FLOAT_T *neuroninputs, FLOAT_T *nextrand, FLOAT_T t, 
							size_t num_neurons, su_mpi_trialparams *tp)
{
	unsigned int num = 0, k;
	for (k=0; k<num_neurons; k++) {
		if (nextrand[k] < t) {
			//neuroninputs[k] += p->randspikesize * (p->fs/1000); 
			neuroninputs[k] += tp->randspikesize;
			nextrand[k] += sk_mpi_expsampl(tp->lambda);
			num += 1;
		}
	}
	return num;
}


/*
 * Function for updating all neurons (calls RK update function above for all
 * nodes).
 */
extern "C"
void sk_mpi_updateneurons(su_mpi_neuron *neurons, FLOAT_T *neuroninputs, IDX_T num_neurons,
						su_mpi_trialparams *tp)
{
	size_t k;
	for (k=0; k<num_neurons; k++) {
		neuronupdate_rk4(&neurons[k].v, &neurons[k].u, neuroninputs[k],
							neurons[k].a, 1000.0/tp->fs);
	}
}

__global__
void sk_mpi_updateneurons_cuker(su_mpi_neuron *neurons, FLOAT_T *neuroninputs, IDX_T num_neurons,
								double fs)
{
	size_t k;
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int stride = blockDim.x * gridDim.x; 
	FLOAT_T K1, K2, K3, K4, L1, L2, L3, L4, half_h, sixth_h, h;
	h = 1000.0/fs;
	half_h = h*0.5;
	sixth_h = h/6.0;

	for (k=index; k<num_neurons; k+=stride) {
		
		K1 = (0.04*(neurons[k].v) + 5.0)*(neurons[k].v) + 140.0 - (neurons[k].u);
		L1 = neurons[k].a*(0.2*(neurons[k].v) - (neurons[k].u));

		K2 = (0.04*( neurons[k].v + half_h*K1 ) + 5.0)*( neurons[k].v +
				half_h*K1 ) + 140.0 - (neurons[k].u + half_h*L1);
		L2 = neurons[k].a*(0.2*( neurons[k].v + half_h*K1 ) - (neurons[k].u + half_h*L1));

		K3 = (0.04*( neurons[k].v + half_h*K2 ) + 5.0)*( neurons[k].v +
				half_h*K2 ) + 140.0 - (neurons[k].u + half_h*L2);
		L3 = neurons[k].a*(0.2*( neurons[k].v + half_h*K2 ) - (neurons[k].u + half_h*L2));

		K4 = (0.04*( neurons[k].v + h*K3 ) + 5.0)*( neurons[k].v + h*K3 ) +
				140.0 - (neurons[k].u + h*L3);
		L4 = neurons[k].a*(0.2*( neurons[k].v + h*K3 ) - ( neurons[k].u + h*L3 ));

		neurons[k].v = neurons[k].v + sixth_h * (K1 + 2*K2 + 2*K3 + K4) + neuroninputs[k];
		neurons[k].u = neurons[k].u + sixth_h * (L1 + 2*L2 + 2*L3 + L4); 
	}
}

extern "C"
void sk_mpi_updateneurons_cuda(su_mpi_neuron *neurons, FLOAT_T *neuroninputs, IDX_T num_neurons,
							   double fs)
{
	unsigned int numblocks = (num_neurons + TPB - 1) / TPB;
	sk_mpi_updateneurons_cuker<<<numblocks, TPB>>>(neurons, neuroninputs, num_neurons, fs);
	hipDeviceSynchronize();
}

extern "C"
unsigned int sk_mpi_checkspiking(su_mpi_neuron *neurons, FLOAT_T *neuronoutputs,
									unsigned int n, FLOAT_T t, spikerecord *sr,
									unsigned int offset)
{
	size_t k;
	unsigned int numspikes=0;
	for (k=0; k<n; k++) {
		neuronoutputs[k] = 0.0;
		if (neurons[k].v >= 30.0) {
			sr_save_spike(sr, k+offset, t);
			neuronoutputs[k] = 1.0;
			neurons[k].v = -65.0;
			neurons[k].u += neurons[k].d;
			numspikes += 1;
		}
	}
	return numspikes;
}

extern "C"
void sk_mpi_updatesynapsetraces(FLOAT_T *traces_syn, FLOAT_T *spike_pre,
								dn_mpi_delaynet *dn, FLOAT_T dt,
								FLOAT_T tau_pre)
{
	size_t k, j;
	FLOAT_T *neuroninputs;

	for (k=0; k<dn->num_nodes_l; k++) {
		for (j=0; j < dn->nodes[k].num_in; j++) {
			neuroninputs = dn_mpi_getinputaddress(k,dn);
			spike_pre[dn->nodes[k].idx_outbuf +j] = neuroninputs[j];
			traces_syn[dn->nodes[k].idx_outbuf +j] = traces_syn[dn->nodes[k].idx_outbuf +j]*(1.0 - (dt/tau_pre)) +
				spike_pre[dn->nodes[k].idx_outbuf +j];
		}
	}
}

__global__
void sk_mpi_updatesynapsetraces_cuker(FLOAT_T *traces_syn, FLOAT_T *spike_pre,
									 dn_mpi_delaynet *dn, FLOAT_T dt,
									 FLOAT_T tau_pre) {
	unsigned int i, j;
	FLOAT_T *neuroninputs;
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int stride = blockDim.x * gridDim.x; 

	for (i=index; i<dn->num_nodes_l; i += stride) {
		for (j=0; j < dn->nodes[i].num_in; j++) {
			neuroninputs = &dn->outputs[dn->nodes[index].idx_outbuf]; 
			spike_pre[dn->nodes[i].idx_outbuf +j] = neuroninputs[j];
			traces_syn[dn->nodes[i].idx_outbuf +j] = traces_syn[dn->nodes[i].idx_outbuf +j]*(1.0 - (dt/tau_pre)) +
				spike_pre[dn->nodes[i].idx_outbuf +j];
		}
	}

}

extern "C"
void sk_mpi_updatesynapsetraces_cuda(FLOAT_T *traces_syn, FLOAT_T *spike_pre,
									 dn_mpi_delaynet *dn, FLOAT_T dt,
									 FLOAT_T tau_pre) {
	unsigned int numblocks = (dn->num_nodes_l + TPB - 1) / TPB;		

	sk_mpi_updatesynapsetraces_cuker<<<numblocks, TPB>>>(traces_syn,
														 spike_pre,
														 dn,
														 dt,
														 tau_pre);
	hipDeviceSynchronize();
}



extern "C"
void sk_mpi_updateneurontraces(FLOAT_T *traces_neu, FLOAT_T *neuronoutputs, IDX_T n,
								FLOAT_T dt, su_mpi_modelparams *mp) 
{
	size_t k;
	for (k=0; k<n; k++) { 		
		traces_neu[k] = traces_neu[k]*(1.0 - (dt/mp->tau_post)) + neuronoutputs[k];
	}
}

__global__
void sk_mpi_updateneurontraces_cuker(FLOAT_T *traces_neu, FLOAT_T *neuronoutputs, IDX_T n,
								FLOAT_T dt, FLOAT_T tau_post) 
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int stride = blockDim.x * gridDim.x; 

	size_t k;
	for (k=index; k<n; k += stride) { 		
		traces_neu[k] = traces_neu[k]*(1.0 - (dt/tau_post)) + neuronoutputs[k];
	}
}

extern "C"
void sk_mpi_updateneurontraces_cuda(FLOAT_T *traces_neu, FLOAT_T *neuronoutputs, IDX_T n,
								FLOAT_T dt, FLOAT_T tau_post) 
{
	unsigned int numblocks = (n + TPB - 1)/TPB;
	sk_mpi_updateneurontraces_cuker<<<numblocks, TPB>>>(traces_neu, neuronoutputs, n, dt, tau_post);

	hipDeviceSynchronize();
}

extern "C"
void sk_mpi_updatesynapses(FLOAT_T *synapses, FLOAT_T *traces_syn, FLOAT_T *traces_neu, 
							FLOAT_T *neuronoutputs, dn_mpi_delaynet *dn, 
							FLOAT_T dt, su_mpi_modelparams *mp)
{
	size_t k, j;
	FLOAT_T *synapseoutputs = dn->outputs;
	for (k=0; k<dn->num_nodes_l; k++) 
	for (j=0; j < dn->nodes[k].num_in; j++) {
		if (synapses[dn->nodes[k].idx_outbuf+j] > 0) { 	// only update excitatory synapses
			synapses[dn->nodes[k].idx_outbuf+j] = synapses[dn->nodes[k].idx_outbuf+j] +
					dt * (mp->a_post * traces_syn[dn->nodes[k].idx_outbuf+j] * neuronoutputs[k] -
						  mp->a_pre * traces_neu[k] * synapseoutputs[dn->nodes[k].idx_outbuf+j]);
			/* clamp value	*/
			synapses[dn->nodes[k].idx_outbuf+j] = synapses[dn->nodes[k].idx_outbuf+j] < 0.0 ? 
										0.0 : synapses[dn->nodes[k].idx_outbuf+j];
			synapses[dn->nodes[k].idx_outbuf+j] = synapses[dn->nodes[k].idx_outbuf+j] > mp->synmax ?
										mp->synmax : synapses[dn->nodes[k].idx_outbuf+j];
		}
	}
}

__global__
void sk_mpi_updatesynapses_cuker(FLOAT_T *synapses, FLOAT_T *traces_syn, FLOAT_T *traces_neu, 
								 FLOAT_T *neuronoutputs, dn_mpi_delaynet *dn, 
								 FLOAT_T dt, FLOAT_T a_pre, FLOAT_T a_post,
								 FLOAT_T synmax)
{
	size_t i, j;
	FLOAT_T *synapseoutputs = dn->outputs;

	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int stride = blockDim.x * gridDim.x; 

	for (i=index; i<dn->num_nodes_l; i += stride) 
	for (j=0; j < dn->nodes[i].num_in; j++) {
		if (synapses[dn->nodes[i].idx_outbuf+j] > 0) { 	// only update excitatory synapses
			synapses[dn->nodes[i].idx_outbuf+j] = synapses[dn->nodes[i].idx_outbuf+j] +
					dt * (a_post * traces_syn[dn->nodes[i].idx_outbuf+j] * neuronoutputs[i] -
						  a_pre * traces_neu[i] * synapseoutputs[dn->nodes[i].idx_outbuf+j]);
			/* clamp value	*/
			synapses[dn->nodes[i].idx_outbuf+j] = synapses[dn->nodes[i].idx_outbuf+j] < 0.0 ? 
										0.0 : synapses[dn->nodes[i].idx_outbuf+j];
			synapses[dn->nodes[i].idx_outbuf+j] =
				synapses[dn->nodes[i].idx_outbuf+j] > synmax ? synmax : synapses[dn->nodes[i].idx_outbuf+j];
		}
	}
}

extern "C"
void sk_mpi_updatesynapses_cuda(FLOAT_T *synapses, FLOAT_T *traces_syn, FLOAT_T *traces_neu, 
								FLOAT_T *neuronoutputs, dn_mpi_delaynet *dn, 
								FLOAT_T dt, FLOAT_T a_pre, FLOAT_T a_post,
								FLOAT_T synmax) 
{
	unsigned int numblocks = (dn->num_nodes_l + TPB - 1)/TPB;

	sk_mpi_updatesynapses_cuker<<<numblocks, TPB>>>(synapses, traces_syn,
			traces_neu, neuronoutputs, dn, dt, a_pre, a_post, synmax);

	hipDeviceSynchronize();
}
