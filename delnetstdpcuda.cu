#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


#include "delnet.h"


/*************************************************************
 *  Macros
 *************************************************************/
//#define SPIKE_BLOCK_SIZE 32768
#define SPIKE_BLOCK_SIZE 8192
#define TPB 256


/*************************************************************
 *  Globals
 *************************************************************/
FLOAT_T g_v_default = -65.0;
FLOAT_T g_u_default = -13.0;

FLOAT_T g_a_exc  = 0.02;
FLOAT_T g_d_exc  = 8.0;
FLOAT_T g_w_exc  = 6.0;

FLOAT_T g_a_inh  = 0.1;
FLOAT_T g_d_inh  = 2.0;
FLOAT_T g_w_inh = -5.0;


/*************************************************************
 *  Structs
 *************************************************************/
typedef struct neuron_s {
	FLOAT_T v;
	FLOAT_T u;
	FLOAT_T a;
	FLOAT_T d;
} neuron;

typedef struct spike_s {
	int neuron;
	FLOAT_T time;
} spike;

typedef struct spikeblock_s {
	long max_spikes;
	long num_spikes;
	spike *spikes;
	struct spikeblock_s *next;
} spikeblock;

typedef struct spikerecord_s {
	spikeblock *head;	
} spikerecord;


/*************************************************************
 *  Functions
 *************************************************************/
__global__ void synapse_trace_kernel_cuda(IDX_T n,
										  IDX_T *offsets,
										  IDX_T *nums_in,
										  FLOAT_T *spike_pre,
										  FLOAT_T *trace_pre, 
										  FLOAT_T *neuroninputs,
										  FLOAT_T dt,
										  FLOAT_T tau_pre) 
{
	unsigned int i, j;
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int stride = blockDim.x * gridDim.x; 

	for (i=index; i < n; i += stride) {
		for (j=0; j < nums_in[i]; j++) {
			spike_pre[offsets[i]+j] = neuroninputs[offsets[i]+j];
			trace_pre[offsets[i]+j] =
				trace_pre[offsets[i]+j]*(1.0 - (dt/tau_pre)) +
							  spike_pre[offsets[i]+j];
		}
	}
}

void synapse_trace_update_cuda(IDX_T n_nodes,
							   IDX_T n_inputs,
							   IDX_T *offsets,
							   IDX_T *nums_in,
							   FLOAT_T *spike_pre,
							   FLOAT_T *trace_pre, 
							   FLOAT_T *neuroninputs,
							   FLOAT_T dt,
							   FLOAT_T tau_pre) 
{
	unsigned int numblocks = (n_nodes + TPB - 1) / TPB;		
	IDX_T *d_offsets=0, *d_nums_in=0;
	FLOAT_T *d_spike_pre=0, *d_trace_pre=0, *d_neuroninputs=0;

	/* move data to GPU */
	hipMemcpy(d_offsets, offsets, n_nodes, hipMemcpyHostToDevice);
	hipMemcpy(d_nums_in, nums_in, n_nodes, hipMemcpyHostToDevice);
	hipMemcpy(d_spike_pre, spike_pre, n_inputs, hipMemcpyHostToDevice);
	hipMemcpy(d_trace_pre, trace_pre, n_inputs, hipMemcpyHostToDevice);
	hipMemcpy(d_neuroninputs, neuroninputs, n_inputs, hipMemcpyHostToDevice);

	synapse_trace_kernel_cuda<<<numblocks, TPB>>>(n_nodes,
												  d_offsets,
												  nums_in,
												  spike_pre,
												  trace_pre,
												  neuroninputs,
												  dt,
												  tau_pre);

	/* move data back from GPU to main memory */
	hipMemcpy(offsets, d_offsets, n_nodes, hipMemcpyDeviceToHost);
	hipMemcpy(nums_in, d_nums_in, n_nodes, hipMemcpyDeviceToHost);
	hipMemcpy(spike_pre, d_spike_pre, n_inputs, hipMemcpyDeviceToHost);
	hipMemcpy(trace_pre, d_trace_pre, n_inputs, hipMemcpyDeviceToHost);
	hipMemcpy(neuroninputs, d_neuroninputs, n_inputs, hipMemcpyDeviceToHost);
}

static inline void synapse_trace_update(IDX_T n,
										IDX_T *offsets,
										IDX_T *nums_in,
										FLOAT_T *spike_pre,
										FLOAT_T *trace_pre, 
										FLOAT_T *neuroninputs,
										FLOAT_T dt,
										FLOAT_T tau_pre) 
{
	IDX_T k,j;

	for (k=0; k<n; k++) {
		for (j=0; j < nums_in[k]; j++) {
			spike_pre[offsets[k]+j] = neuroninputs[offsets[k]+j];
			trace_pre[offsets[k]+j] =
				trace_pre[offsets[k]+j]*(1.0 - (dt/tau_pre)) +
							  spike_pre[offsets[k]+j];
		}
	}
}



static inline void synapse_strength_update(IDX_T n_exc,
										   IDX_T *exc_offset,
										   IDX_T *offsets,
										   IEX_T *nums_in,
										   trace_pre,
										   trace_post,
										   spike_pre,
										   spike_post,
										   synapses,
										   dt,
										   a_pre,
										   a_post,
										   synmax)
{
}

double dd_sum_double(double *vals, size_t n) {
	double sum = 0.0;
	for (size_t k=0; k<n; k++) 
		sum += vals[k];
	return sum;
}

double dd_avg_double(double *vals, size_t n) {
	double sum = dd_sum_double(vals, n);
	return sum / ((double) n);
}

spikerecord *sr_init()
{
	spikerecord *rec;
	rec = (spikerecord *) malloc(sizeof(spikerecord));
	rec->head = (spikeblock *) malloc(sizeof(spikeblock));
	rec->head->max_spikes = SPIKE_BLOCK_SIZE;
	rec->head->num_spikes = 0;
	rec->head->spikes = (spike *) malloc(sizeof(spike)*SPIKE_BLOCK_SIZE);
	rec->head->next = 0;

	return rec;
}

void sr_save_spike(spikerecord *sr, int neuron, FLOAT_T time)
{
	if (sr->head->num_spikes < sr->head->max_spikes) {
		sr->head->spikes[sr->head->num_spikes].neuron = neuron;
		sr->head->spikes[sr->head->num_spikes].time = time;
		sr->head->num_spikes += 1;
	}
	else {
		/* allocate new spike block and saves spike */
		spikeblock *newest = (spikeblock *) malloc(sizeof(spikeblock));
		newest->max_spikes = SPIKE_BLOCK_SIZE;
		newest->num_spikes = 0;
		newest->spikes = (spike *) malloc(sizeof(spike)*SPIKE_BLOCK_SIZE);
		newest->next = sr->head;
		sr->head = newest;
		sr->head->spikes[sr->head->num_spikes].neuron = neuron;
		sr->head->spikes[sr->head->num_spikes].time = time;
		sr->head->num_spikes += 1;
	}
}


/*
 * Revise this later so that spikes are in order (they are in order
 * by block, but blocks are reversed)
 */
spike *sr_spike_summary(spikerecord *sr)
{
	/* Calculate total number of spikes and allocate */
	long num_spikes = 0;
	spike *spikes_all;
	spikeblock *curblock = sr->head;
	while (curblock != 0) {
		num_spikes += curblock->num_spikes;
		curblock = curblock->next;
	}

	spikes_all = (spike *) malloc(sizeof(spike)*num_spikes);

	curblock = sr->head;
	long idx = 0;
	while (curblock != 0) {
		for (int i=0; i < curblock->num_spikes; i++) {
			spikes_all[idx] = curblock->spikes[i];
			idx += 1;
		}
		curblock = curblock->next;
	}
	return spikes_all;
}

void sr_free(spikerecord *sr)
{
	spikeblock *curblock = sr->head;
	spikeblock *newest;
	while (curblock != 0) {
		free(curblock->spikes);
		newest = curblock->next;
		free(curblock);
		curblock = newest;
	}
	free(sr);
}

/*
 * -------------------- Util Functions --------------------
 */

dn_list_uint *dn_list_uint_init() {
	dn_list_uint *newlist;
	newlist = (dn_list_uint *)malloc(sizeof(dn_list_uint));
	newlist->count = 0;
	newlist->head = NULL; 

	return newlist;
}

void dn_list_uint_push(dn_list_uint *l, unsigned int val) {
	dn_listnode_uint *newnode;
	newnode = (dn_listnode_uint *)malloc(sizeof(dn_listnode_uint));
	newnode->val = val;
	newnode->next = l->head;
	l->head = newnode;
	l->count += 1;
}

unsigned int dn_list_uint_pop(dn_list_uint *l) {
	unsigned int val;
	dn_listnode_uint *temp;

	if (l->head != NULL) {
		val = l->head->val;
		temp = l->head;
		l->head = l->head->next;
		free(temp);
		l->count -= 1;
	}
	else {
		// trying to pop an empty list
		//val = 0;
		exit(-1);
	}
	return val;
}

void dn_list_uint_free(dn_list_uint *l) {
	while (l->head != NULL) {
		dn_list_uint_pop(l);
	}
	free(l);
}


dn_vec_float dn_orderbuf(IDX_T which, dn_delaynet *dn) {
	IDX_T k, n, idx;
	dn_vec_float output;
	
	n = dn->del_lens[which];
	output.n = n;
	output.data = (FLOAT_T *)malloc(sizeof(FLOAT_T) * n);

	for (k=0; k<n; k++) {
		idx = dn->del_startidces[which] +
			((dn->del_offsets[which]+k) % dn->del_lens[which]);
		output.data[n-k-1] = dn->delaybuf[idx];
	}
	return output;
}

char *dn_vectostr(dn_vec_float input) {
	int k;
	char *output;
	output = (char *) malloc(sizeof(char)*(input.n+1));
	output[input.n] = '\0';
	for(k=0; k < input.n; k++) {
		output[k] = input.data[k] == 0.0 ? '-' : '1';
	}
	return output;
}



/*
 * -------------------- delnet Functions --------------------
 */

void dn_pushoutput(FLOAT_T val, IDX_T idx, dn_delaynet *dn) 
{
	IDX_T i1, i2, k;

	i1 = dn->nodes[idx].idx_io;
	i2 = i1 + dn->nodes[idx].num_out;

	for (k = i1; k < i2; k++)
		dn->inputs[k] = val;
}


/* No getinputs()... would need to return vector */
dn_vec_float dn_getinputvec(dn_delaynet *dn) {
	dn_vec_float inputs;	
	inputs.data = (FLOAT_T *) malloc(sizeof(FLOAT_T)*dn->num_delays);
	inputs.n = dn->num_delays;
	for (int i=0; i<dn->num_delays; i++) {
		inputs.data[i] = dn->inputs[i];
	}
	return inputs;
}

/* get inputs to neurons (outputs of delaynet)... */
FLOAT_T *dn_getinputaddress(IDX_T idx, dn_delaynet *dn) {
	return &dn->outputs[dn->nodes[idx].idx_oi];
}


void dn_advance(dn_delaynet *dn)
{
	IDX_T k;

	for(k=0; k < dn->num_delays; k++) {
		dn->delaybuf[dn->del_startidces[k] + dn->del_offsets[k]] =
															dn->inputs[k];
	}

	for(k=0; k < dn->num_delays; k++) {
		dn->del_offsets[k] = (dn->del_offsets[k] + 1) % dn->del_lens[k];
	}

	for (k=0; k < dn->num_delays; k++) {
		dn->outputs[dn->inverseidx[k]] =
				dn->delaybuf[dn->del_startidces[k]+dn->del_offsets[k]];
	}
}


unsigned int *dn_blobgraph(unsigned int n, float p, unsigned int maxdel) {
	unsigned int count = 0;
	unsigned int *delmat;
	unsigned int i, j;
	delmat = (unsigned int *) malloc(sizeof(unsigned int)*n*n);

	for (i=0; i<n; i++) 
	for (j=0; j<n; j++) {
		if (unirand() < p && i != j) {
			delmat[i*n+j] = getrandom(maxdel) + 1;
			count += 1;
		}
		else 
			delmat[i*n+j] = 0;
	}

	return delmat;
}


dn_delaynet *dn_delnetfromgraph(unsigned int *g, unsigned int n) {
	unsigned int i, j, delcount, startidx;
	unsigned int deltot, numlines;
	dn_delaynet *dn;
	dn_list_uint **nodes_in;

	dn = (dn_delaynet *) malloc(sizeof(dn_delaynet));
	nodes_in = (dn_list_uint **) malloc(sizeof(dn_list_uint *)*n);
	for (i=0; i<n; i++)
		nodes_in[i] = dn_list_uint_init();

	deltot = 0;
	numlines = 0;
	for (i=0; i<n*n; i++) {
		deltot += g[i];
		numlines += g[i] != 0 ? 1 : 0;
	}
	dn->num_delays = numlines;
	dn->buf_len = deltot;
	dn->num_nodes = n;

	dn->delaybuf = (FLOAT_T *) calloc(deltot, sizeof(FLOAT_T));
	dn->inputs   = (FLOAT_T *) calloc(numlines, sizeof(FLOAT_T));
	dn->outputs  = (FLOAT_T *) calloc(numlines, sizeof(FLOAT_T));

	dn->del_offsets 	= (IDX_T *) malloc(sizeof(IDX_T)*numlines);
	dn->del_startidces 	= (IDX_T *) malloc(sizeof(IDX_T)*numlines);
	dn->del_lens 		= (IDX_T *) malloc(sizeof(IDX_T)*numlines);
	dn->del_sources 	= (IDX_T *) malloc(sizeof(IDX_T)*numlines);
	dn->del_targets 	= (IDX_T *) malloc(sizeof(IDX_T)*numlines);
	dn->nodes 			= (dn_node *) malloc(sizeof(dn_node)*n);

	/* init nodes */
	for (i=0; i<n; i++) {
		dn->nodes[i].idx_oi = 0;
		dn->nodes[i].num_in = 0;
		dn->nodes[i].idx_io = 0;
		dn->nodes[i].num_out = 0;
	}

	/* work through graph, allocate delay lines */
	delcount = 0;
	startidx = 0;
	for (i = 0; i<n; i++)
	for (j = 0; j<n; j++) {
		if (g[i*n + j] != 0) {
			dn_list_uint_push(nodes_in[j], i);

			dn->del_offsets[delcount] = 0;
			dn->del_startidces[delcount] = startidx;
			dn->del_lens[delcount] = g[i*n+j];
			dn->del_sources[delcount] = i;
			dn->del_targets[delcount] = j;

			dn->nodes[i].num_out += 1;

			startidx += g[i*n +j];
			delcount += 1;
		}
	}
	
	/* work out rest of index arithmetic */
	unsigned int *num_outputs, *in_base_idcs;
	num_outputs  = (unsigned int *) calloc(n, sizeof(unsigned int));
	in_base_idcs = (unsigned int *) calloc(n, sizeof(unsigned int));
	for (i=0; i<n; i++) {
		num_outputs[i] = dn->nodes[i].num_out;
		for (j=0; j<i; j++)
			in_base_idcs[i] += num_outputs[j]; 	// check logic here
		//in_base_idcs[i] = i == 0 ? 0 : in_base_idcs[i-1] + num_outputs[i];
	}

	unsigned int idx = 0;
	for (i=0; i<n; i++) {
		dn->nodes[i].num_in = nodes_in[i]->count;
		dn->nodes[i].idx_oi = idx;
		idx += dn->nodes[i].num_in;
		dn->nodes[i].idx_io = in_base_idcs[i];
	}

	unsigned int *num_inputs, *out_base_idcs, *out_counts, *inverseidces;
	num_inputs 		= (unsigned int *) calloc(n, sizeof(unsigned int));
	out_base_idcs 	= (unsigned int *) calloc(n, sizeof(unsigned int));
	out_counts 		= (unsigned int *) calloc(n, sizeof(unsigned int));
	for (i=0; i<n; i++) {
		num_inputs[i] = dn->nodes[i].num_in;
		for (j=0; j<i; j++)
			out_base_idcs[i] += num_inputs[j]; // check logic here
		//out_base_idcs[i] = i == 0 ? 0 : in_base_idcs[i-1] + num_inputs[i];
	}

	inverseidces = (unsigned int *) calloc(numlines, sizeof(unsigned int));
	for (i=0; i < numlines; i++) {
		inverseidces[i] = out_base_idcs[dn->del_targets[i]] + 
						  out_counts[dn->del_targets[i]];
		out_counts[dn->del_targets[i]] += 1;
	}
	dn->inverseidx = inverseidces;

	/* Clean up */
	for (i=0; i<n; i++)
		dn_list_uint_free(nodes_in[i]);
	free(nodes_in);
	free(num_outputs);
	free(in_base_idcs);
	free(num_inputs);
	free(out_base_idcs);
	free(out_counts);

	return dn;
}

void dn_freedelnet(dn_delaynet *dn) {
	free(dn->del_offsets);
	free(dn->del_startidces);
	free(dn->del_lens);
	free(dn->del_sources);
	free(dn->del_targets);
	free(dn->inputs);
	free(dn->outputs);
	free(dn->inverseidx);
	free(dn->delaybuf);
	free(dn->nodes);
	free(dn);
}



/*************************************************************
 *  Main
 *************************************************************/

/**
 * @brief Simulation with delnet library
 *
 */
int main(int argc, char *argv[])
{
	FLOAT_T fs, dur, dt, t;
	FLOAT_T tau_pre, tau_post, a_pre, a_post, synbump, synmax;
	unsigned int i, j, k, numsteps;
	unsigned int n, n_exc, n_inh;
	unsigned int *g;
	unsigned long int numspikes = 0;
	float p_contact;
	dn_delaynet *dn;
	spikerecord *sr = sr_init();
	clock_t t_start, t_finish;
	double *gettinginputs, *updatingsyntraces, *updatingneurons, *spikechecking,
			*updatingneutraces, *updatingsynstrengths, *pushingoutput,
			*advancingbuffer;

	if (argc != 2)
		n = 1000;
	else
		n = atoi(argv[1]);


	/* trial parameters */
	fs = 1000.0;
	dur = 2.0;
	p_contact = 0.1;
	//n = 2000;
	tau_pre = 0.02;
	tau_post = 0.02;
	a_pre = 0.12;
	a_post = 0.1;
	synbump = 0.00000;
	synmax = 10.0;



	/* derived parameters */
	n_exc = n*0.8;
	n_inh = n*0.2;
	n = n_exc + n_inh;  // in case of rounding issue
	dt = 1.0/fs;
	numsteps = dur/dt;

	/* print trial parameters */
	printf("Sampling Frequency: \t%f\n", fs);
	printf("Duration: \t\t%f\n", dur);
	printf("Number of nodes: \t%d\n", n);
	printf(" 	Excitatory: \t%d\n", n_exc);
	printf(" 	Inhibitory: \t%d\n", n_inh);
	printf("Probability of contact:\t%f\n", p_contact);
	printf("tau_pre:\t\t%f\n", tau_pre);
	printf("A_pre:\t\t\t%f\n", a_pre);
	printf("tau_post:\t\t%f\n", tau_pre);
	printf("A_post:\t\t\t%f\n", a_pre);
	printf("----------------------------------------\n");

	/* set up graph */
	g = dn_blobgraph(n, p_contact, 20);
	for (i=n_exc; i<n; i++) 			// only last 200 rows
	for (j=0; j<n; j++) { 				
		g[i*n+j] = g[i*n+j] != 0 ? 1 : 0; 	// 1 ms delay for inh
	}

	/* generate delay network */
	dn = dn_delnetfromgraph(g, n);

	/* initialize neuron and synapse state  */
	neuron *neurons 	= (neuron *) malloc(sizeof(neuron)*n);
	FLOAT_T *trace_post = (FLOAT_T *) calloc(n_exc, sizeof(FLOAT_T));
	FLOAT_T *spike_post = (FLOAT_T *) calloc(n_exc, sizeof(FLOAT_T));
	IDX_T *offsets 		= (IDX_T *) malloc(sizeof(IDX_T)*n);
	FLOAT_T *trace_pre; 	// pack this
	FLOAT_T *spike_pre; 	// and following
	FLOAT_T *synapses; 		// for speed?

	unsigned long cum_in = 0, exc_offset;
	for (i=0; i<n_exc; i++) {
		neurons[i].v = g_v_default;
		neurons[i].u = g_u_default;
		neurons[i].a = g_a_exc;
		neurons[i].d = g_d_exc;
		offsets[i] = cum_in;
		cum_in += dn->nodes[i].num_in;
	}
	exc_offset = cum_in;

	for (i=n_exc; i<n; i++) {
		neurons[i].v = g_v_default;
		neurons[i].u = g_u_default;
		neurons[i].a = g_a_inh;
		neurons[i].d = g_d_inh;
		offsets[i] = cum_in;
		cum_in += dn->nodes[i].num_in;
	}

	trace_pre = (FLOAT_T *) calloc(exc_offset, sizeof(FLOAT_T));
	spike_pre = (FLOAT_T *) calloc(exc_offset, sizeof(FLOAT_T));
	synapses  = (FLOAT_T *) calloc(cum_in, sizeof(FLOAT_T));
	for (i=0; i<n_exc; i++)
		synapses[i] = g_w_exc;
	for (; i<n; i++)
		synapses[i] = g_w_inh;


	/* for profiling */	
	gettinginputs 		 = (double *) malloc(sizeof(double)*numsteps);
	updatingsyntraces 	 = (double *) malloc(sizeof(double)*numsteps);
	updatingneurons 	 = (double *) malloc(sizeof(double)*numsteps);
	spikechecking 		 = (double *) malloc(sizeof(double)*numsteps);
	updatingneutraces 	 = (double *) malloc(sizeof(double)*numsteps);
	updatingsynstrengths = (double *) malloc(sizeof(double)*numsteps);
	pushingoutput 		 = (double *) malloc(sizeof(double)*numsteps);
	advancingbuffer 	 = (double *) malloc(sizeof(double)*numsteps);

	/* intermediate variables for simulation -- clean these up later*/
	FLOAT_T *neuroninputs, *invals, *outvals;
	IDX_T *nums_in;
	invals  = (FLOAT_T *) calloc(n, sizeof(FLOAT_T));
	outvals = (FLOAT_T *) calloc(n, sizeof(FLOAT_T));
	nums_in = (IDX_T *) calloc(n, sizeof(IDX_T));
	for (i=0; i<n; i++) 
		nums_in[i] = dn->nodes[i].num_in; 	// see if helps speed


	/* start simulation */
	for (i=0; i<numsteps; i++) {

		/* print updates */
		t = dt*i;
		if (i%1000 == 0)
			printf("Time: %f\n", t);


		/* get inputs to neuron */		
		t_start = clock();
		for (k=0; k<n; k++) {
			neuroninputs = dn_getinputaddress(k, dn);

			/* weighted sum */
			for (j=0; j < nums_in[k]; j++)
				invals[k] += *(neuroninputs+j) * synapses[offsets[k]+j];

			/* added noise */
			if (unirand() < 1.0/n)
				invals[k] += 20.0 * (fs/1000.0);
		}
		t_finish = clock();
		gettinginputs[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* update synapse traces */
		t_start = clock();
	    synapse_trace_update_cuda(n_exc, 		// number of excitatory neurons
	  						      exc_offset, 	// number of excitatory synapses
	  						      offsets,
	  						      nums_in,
	  						      spike_pre,
	  						      trace_pre, 
	  						      neuroninputs,
	  						      dt,
	  						      tau_pre);
		t_finish = clock();
		updatingsyntraces[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* update neuron state */
		t_start = clock();
		for (k=0; k<n; k++) {
			neurons[k].v += 500.0 * dt * (( 0.04 * neurons[k].v + 5.0) *
							neurons[k].v + 140.0 - neurons[k].u + invals[k]);
			neurons[k].v += 500.0 * dt * (( 0.04 * neurons[k].v + 5.0) *
							neurons[k].v + 140.0 - neurons[k].u + invals[k]);
			neurons[k].u += 1000.0 * dt * neurons[k].a *
								(0.2 * neurons[k].v - neurons[k].u);
			invals[k] = 0.0;
		}
		t_finish = clock();
		updatingneurons[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;

		/* check if spiked and calculate output */
		t_start = clock();
		for (k=0; k<n; k++) {
			outvals[k] = 0.0;
			if (neurons[k].v >= 30.0) {
				sr_save_spike(sr, k, t);
				outvals[k] = 1.0;
				neurons[k].v = -65.0;
				neurons[k].u += neurons[k].d;
				numspikes += 1;
			}
		}
		t_finish = clock();
		spikechecking[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;

		/* update neuron trace */		
		t_start = clock();
		for (k=0; k<n; k++) {
			if (k < n_exc) {
				spike_post[k] = outvals[k];
				trace_post[k] = trace_post[k]*(1.0 - (dt/tau_post)) +
								spike_post[k];
			}
		}
		t_finish = clock();
		updatingneutraces[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* update synapse strengths */
		t_start = clock();
		synapse_strength_update(n_exc,
							    exc_offset,
								offsets,
								nums_in,
								trace_pre,
								trace_post,
								spike_pre,
								spike_post,
								synapses,
								dt,
								a_pre,
								a_post,
								synmax);
		/*
		for (k=0; k<n_exc; k++) {
			for (j=0; j < nums_in[k]; j++) {
				synapses[offsets[k]+j] = synapses[offsets[k]+j] + synbump +
						dt * (a_post * trace_pre[offsets[k]+j] * spike_post[k] -
							  a_pre * trace_post[k] * spike_pre[offsets[k]+j]);
				synapses[offsets[k]+j] =
					synapses[offsets[k]+j] < 0.0 ? 0.0 : synapses[offsets[k]+j];
				synapses[offsets[k]+j] =
					synapses[offsets[k]+j] > synmax ? synmax : synapses[offsets[k]+j];
				//spike_post[k] = 0.0;
			}
		}
		*/
		t_finish = clock();
		updatingsynstrengths[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* push the output into the buffer */
		t_start = clock();
		for (k=0; k<n; k++)
			dn_pushoutput(outvals[k], k, dn);
		t_finish = clock();
		pushingoutput[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* advance the buffer */
		t_start = clock();
		dn_advance(dn);
		t_finish = clock();
		advancingbuffer[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;
	}

	/* performance analysis */
	double cycletime, cumtime = 0.0;

	printf("----------------------------------------\n");

	cycletime = 1000.0*dd_sum_double(gettinginputs, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Getting inputs:\t\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(updatingsyntraces, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Update syntraces:\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(updatingneurons, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Update neurons:\t\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(spikechecking, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Check spiked:\t\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(updatingneutraces, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Update neurtrace:\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(updatingsynstrengths, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Update syn strength:\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(pushingoutput, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Pushing buffer:\t\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(advancingbuffer, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Advancing buffer:\t %f (ms)\n", cycletime);

	printf("Total cycle time:\t %f (ms)\n", cumtime);
	printf("Time per second: \t %f (ms)\n", cumtime*fs);

	/* Save spikes */
	FILE *spike_file;
	spike_file = fopen( "delnetstdp.dat", "w" );
	spike *firings = sr_spike_summary(sr);
	for (i=0; i<numspikes; i++)
		fprintf(spike_file, "%f  %d\n", firings[i].time, firings[i].neuron);
	fclose(spike_file);


	/* Clean up */
	dn_freedelnet(dn);
	sr_free(sr);
	free(g);
	free(firings);
	free(trace_post);
	free(spike_post);
	free(neurons);
	free(trace_pre);
	free(spike_pre);
	free(synapses);
	free(gettinginputs);
	free(updatingsyntraces);
	free(updatingneurons);
	free(spikechecking);
	free(updatingneutraces);
	free(updatingsynstrengths);
	free(pushingoutput);
	free(advancingbuffer);

	return 0;
}
