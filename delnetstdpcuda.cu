#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "delnet.h"


/*************************************************************
 *  Macros
 *************************************************************/
//#define SPIKE_BLOCK_SIZE 32768
#define SPIKE_BLOCK_SIZE 8192
#define TPB 256


/*************************************************************
 *  Globals
 *************************************************************/
FLOAT_T g_v_default = -65.0;
FLOAT_T g_u_default = -13.0;

FLOAT_T g_a_exc  = 0.02;
FLOAT_T g_d_exc  = 8.0;
FLOAT_T g_w_exc  = 6.0;

FLOAT_T g_a_inh  = 0.1;
FLOAT_T g_d_inh  = 2.0;
FLOAT_T g_w_inh = -5.0;


/*************************************************************
 *  Structs
 *************************************************************/
typedef struct neuron_s {
	FLOAT_T v;
	FLOAT_T u;
	FLOAT_T a;
	FLOAT_T d;
} neuron;

typedef struct spike_s {
	int neuron;
	FLOAT_T time;
} spike;

typedef struct spikeblock_s {
	long max_spikes;
	long num_spikes;
	spike *spikes;
	struct spikeblock_s *next;
} spikeblock;

typedef struct spikerecord_s {
	spikeblock *head;	
} spikerecord;


/*************************************************************
 *  Functions
 *************************************************************/
__global__ void synapse_trace_kernel_cuda(IDX_T n,
										  IDX_T *offsets,
										  IDX_T *nums_in,
										  FLOAT_T *spike_pre,
										  FLOAT_T *trace_pre, 
										  FLOAT_T *neuroninputs,
										  FLOAT_T dt,
										  FLOAT_T tau_pre) 
{
	unsigned int i, j;
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int stride = blockDim.x * gridDim.x; 

	for (i=index; i < n; i += stride) {
		for (j=0; j < nums_in[i]; j++) {
			spike_pre[offsets[i]+j] = neuroninputs[offsets[i]+j];
			trace_pre[offsets[i]+j] =
				trace_pre[offsets[i]+j]*(1.0 - (dt/tau_pre)) +
							  spike_pre[offsets[i]+j];
		}
	}
}

void synapse_trace_update_cuda(IDX_T n_nodes,
							   IDX_T n_inputs,
							   IDX_T *offsets,
							   IDX_T *nums_in,
							   FLOAT_T *spike_pre,
							   FLOAT_T *trace_pre, 
							   FLOAT_T *neuroninputs,
							   FLOAT_T dt,
							   FLOAT_T tau_pre) 
{
	unsigned int numblocks = (n_nodes + TPB - 1) / TPB;		
	IDX_T *d_offsets=0, *d_nums_in=0;
	FLOAT_T *d_spike_pre=0, *d_trace_pre=0, *d_neuroninputs=0;

	hipMemcpy(d_offsets, offsets, n_nodes, hipMemcpyHostToDevice);
	hipMemcpy(d_nums_in, nums_in, n_nodes, hipMemcpyHostToDevice);
	hipMemcpy(d_spike_pre, spike_pre, n_inputs, hipMemcpyHostToDevice);
	hipMemcpy(d_trace_pre, trace_pre, n_inputs, hipMemcpyHostToDevice);
	hipMemcpy(d_neuroninputs, neuroninputs, n_inputs, hipMemcpyHostToDevice);

	synapse_trace_kernel_cuda<<<numblocks, TPB>>>(n_nodes,
												  d_offsets,
												  nums_in,
												  spike_pre,
												  trace_pre,
												  neuroninputs,
												  dt,
												  tau_pre);
	hipMemcpy(offsets, d_offsets, n_nodes, hipMemcpyDeviceToHost);
	hipMemcpy(nums_in, d_nums_in, n_nodes, hipMemcpyDeviceToHost);
	hipMemcpy(spike_pre, d_spike_pre, n_inputs, hipMemcpyDeviceToHost);
	hipMemcpy(trace_pre, d_trace_pre, n_inputs, hipMemcpyDeviceToHost);
	hipMemcpy(neuroninputs, d_neuroninputs, n_inputs, hipMemcpyDeviceToHost);
}

static inline void synapse_trace_update(IDX_T n,
										IDX_T *offsets,
										IDX_T *nums_in,
										FLOAT_T *spike_pre,
										FLOAT_T *trace_pre, 
										FLOAT_T *neuroninputs,
										FLOAT_T dt,
										FLOAT_T tau_pre) 
{
	IDX_T k,j;

	for (k=0; k<n; k++) {
		for (j=0; j < nums_in[k]; j++) {
			spike_pre[offsets[k]+j] = neuroninputs[offsets[k]+j];
			trace_pre[offsets[k]+j] =
				trace_pre[offsets[k]+j]*(1.0 - (dt/tau_pre)) +
							  spike_pre[offsets[k]+j];
		}
	}
}




double dd_sum_double(double *vals, size_t n) {
	double sum = 0.0;
	for (size_t k=0; k<n; k++) 
		sum += vals[k];
	return sum;
}

double dd_avg_double(double *vals, size_t n) {
	double sum = dd_sum_double(vals, n);
	return sum / ((double) n);
}

spikerecord *sr_init()
{
	spikerecord *rec;
	rec = (spikerecord *) malloc(sizeof(spikerecord));
	rec->head = (spikeblock *) malloc(sizeof(spikeblock));
	rec->head->max_spikes = SPIKE_BLOCK_SIZE;
	rec->head->num_spikes = 0;
	rec->head->spikes = (spike *) malloc(sizeof(spike)*SPIKE_BLOCK_SIZE);
	rec->head->next = 0;

	return rec;
}

void sr_save_spike(spikerecord *sr, int neuron, FLOAT_T time)
{
	if (sr->head->num_spikes < sr->head->max_spikes) {
		sr->head->spikes[sr->head->num_spikes].neuron = neuron;
		sr->head->spikes[sr->head->num_spikes].time = time;
		sr->head->num_spikes += 1;
	}
	else {
		/* allocate new spike block and saves spike */
		spikeblock *newest = (spikeblock *) malloc(sizeof(spikeblock));
		newest->max_spikes = SPIKE_BLOCK_SIZE;
		newest->num_spikes = 0;
		newest->spikes = (spike *) malloc(sizeof(spike)*SPIKE_BLOCK_SIZE);
		newest->next = sr->head;
		sr->head = newest;
		sr->head->spikes[sr->head->num_spikes].neuron = neuron;
		sr->head->spikes[sr->head->num_spikes].time = time;
		sr->head->num_spikes += 1;
	}
}


/*
 * Revise this later so that spikes are in order (they are in order
 * by block, but blocks are reversed)
 */
spike *sr_spike_summary(spikerecord *sr)
{
	/* Calculate total number of spikes and allocate */
	long num_spikes = 0;
	spike *spikes_all;
	spikeblock *curblock = sr->head;
	while (curblock != 0) {
		num_spikes += curblock->num_spikes;
		curblock = curblock->next;
	}

	spikes_all = (spike *) malloc(sizeof(spike)*num_spikes);

	curblock = sr->head;
	long idx = 0;
	while (curblock != 0) {
		for (int i=0; i < curblock->num_spikes; i++) {
			spikes_all[idx] = curblock->spikes[i];
			idx += 1;
		}
		curblock = curblock->next;
	}
	return spikes_all;
}

void sr_free(spikerecord *sr)
{
	spikeblock *curblock = sr->head;
	spikeblock *newest;
	while (curblock != 0) {
		free(curblock->spikes);
		newest = curblock->next;
		free(curblock);
		curblock = newest;
	}
	free(sr);
}

/*************************************************************
 *  Main
 *************************************************************/

/**
 * @brief Simulation with delnet library
 *
 */
int main()
{
	FLOAT_T fs, dur, dt, t;
	FLOAT_T tau_pre, tau_post, a_pre, a_post, synbump, synmax;
	unsigned int i, j, k, numsteps;
	unsigned int n, n_exc, n_inh;
	unsigned int *g;
	unsigned long int numspikes = 0;
	float p_contact;
	dn_delaynet *dn;
	spikerecord *sr = sr_init();
	clock_t t_start, t_finish;
	double *gettinginputs, *updatingsyntraces, *updatingneurons, *spikechecking,
			*updatingneutraces, *updatingsynstrengths, *pushingoutput,
			*advancingbuffer;

	/* trial parameters */
	fs = 1000.0;
	dur = 2.0;
	p_contact = 0.1;
	n = 1000;
	tau_pre = 0.02;
	tau_post = 0.02;
	a_pre = 0.12;
	a_post = 0.1;
	synbump = 0.00000;
	synmax = 10.0;

	/* derived parameters */
	n_exc = n*0.8;
	n_inh = n*0.2;
	n = n_exc + n_inh;  // in case of rounding issue
	dt = 1.0/fs;
	numsteps = dur/dt;

	/* print trial parameters */
	printf("Sampling Frequency: \t%f\n", fs);
	printf("Duration: \t\t%f\n", dur);
	printf("Number of nodes: \t%d\n", n);
	printf(" 	Excitatory: \t%d\n", n_exc);
	printf(" 	Inhibitory: \t%d\n", n_inh);
	printf("Probability of contact:\t%f\n", p_contact);
	printf("tau_pre:\t\t%f\n", tau_pre);
	printf("A_pre:\t\t\t%f\n", a_pre);
	printf("tau_post:\t\t%f\n", tau_pre);
	printf("A_post:\t\t\t%f\n", a_pre);
	printf("----------------------------------------\n");

	/* set up graph */
	g = dn_blobgraph(n, p_contact, 20);
	for (i=n_exc; i<n; i++) 			// only last 200 rows
	for (j=0; j<n; j++) { 				
		g[i*n+j] = g[i*n+j] != 0 ? 1 : 0; 	// 1 ms delay for inh
	}

	/* generate delay network */
	dn = dn_delnetfromgraph(g, n);

	/* initialize neuron and synapse state  */
	neuron *neurons 	= (neuron *) malloc(sizeof(neuron)*n);
	FLOAT_T *trace_post = (FLOAT_T *) calloc(n_exc, sizeof(FLOAT_T));
	FLOAT_T *spike_post = (FLOAT_T *) calloc(n_exc, sizeof(FLOAT_T));
	IDX_T *offsets 		= (IDX_T *) malloc(sizeof(IDX_T)*n);
	FLOAT_T *trace_pre; 	// pack this
	FLOAT_T *spike_pre; 	// and following
	FLOAT_T *synapses; 		// for speed?

	unsigned long cum_in = 0, exc_offset;
	for (i=0; i<n_exc; i++) {
		neurons[i].v = g_v_default;
		neurons[i].u = g_u_default;
		neurons[i].a = g_a_exc;
		neurons[i].d = g_d_exc;
		offsets[i] = cum_in;
		cum_in += dn->nodes[i].num_in;
	}
	exc_offset = cum_in;

	for (i=n_exc; i<n; i++) {
		neurons[i].v = g_v_default;
		neurons[i].u = g_u_default;
		neurons[i].a = g_a_inh;
		neurons[i].d = g_d_inh;
		offsets[i] = cum_in;
		cum_in += dn->nodes[i].num_in;
	}

	trace_pre = (FLOAT_T *) calloc(exc_offset, sizeof(FLOAT_T));
	spike_pre = (FLOAT_T *) calloc(exc_offset, sizeof(FLOAT_T));
	synapses  = (FLOAT_T *) calloc(cum_in, sizeof(FLOAT_T));
	for (i=0; i<n_exc; i++)
		synapses[i] = g_w_exc;
	for (; i<n; i++)
		synapses[i] = g_w_inh;


	/* for profiling */	
	gettinginputs 		 = (double *) malloc(sizeof(double)*numsteps);
	updatingsyntraces 	 = (double *) malloc(sizeof(double)*numsteps);
	updatingneurons 	 = (double *) malloc(sizeof(double)*numsteps);
	spikechecking 		 = (double *) malloc(sizeof(double)*numsteps);
	updatingneutraces 	 = (double *) malloc(sizeof(double)*numsteps);
	updatingsynstrengths = (double *) malloc(sizeof(double)*numsteps);
	pushingoutput 		 = (double *) malloc(sizeof(double)*numsteps);
	advancingbuffer 	 = (double *) malloc(sizeof(double)*numsteps);

	/* intermediate variables for simulation -- clean these up later*/
	FLOAT_T *neuroninputs, *invals, *outvals;
	IDX_T *nums_in;
	invals  = (FLOAT_T *) calloc(n, sizeof(FLOAT_T));
	outvals = (FLOAT_T *) calloc(n, sizeof(FLOAT_T));
	nums_in = (IDX_T *) calloc(n, sizeof(IDX_T));
	for (i=0; i<n; i++) 
		nums_in[i] = dn->nodes[i].num_in; 	// see if helps speed


	/* start simulation */
	for (i=0; i<numsteps; i++) {

		/* print updates */
		t = dt*i;
		if (i%1000 == 0)
			printf("Time: %f\n", t);


		/* get inputs to neuron */		
		t_start = clock();
		for (k=0; k<n; k++) {
			neuroninputs = dn_getinputaddress(k, dn);

			/* weighted sum */
			for (j=0; j < nums_in[k]; j++)
				invals[k] += *(neuroninputs+j) * synapses[offsets[k]+j];

			/* added noise */
			if (unirand() < 1.0/n)
				invals[k] += 20.0 * (fs/1000.0);
		}
		t_finish = clock();
		gettinginputs[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* update synapse traces */
		t_start = clock();
		synapse_trace_update(n_exc,
							 offsets,
						     nums_in,
							 spike_pre,
							 trace_pre, 
							 dn_getinputaddress(0,dn),
							 dt,
							 tau_pre);
		t_finish = clock();
		updatingsyntraces[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* update neuron state */
		t_start = clock();
		for (k=0; k<n; k++) {
			neurons[k].v += 500.0 * dt * (( 0.04 * neurons[k].v + 5.0) *
							neurons[k].v + 140.0 - neurons[k].u + invals[k]);
			neurons[k].v += 500.0 * dt * (( 0.04 * neurons[k].v + 5.0) *
							neurons[k].v + 140.0 - neurons[k].u + invals[k]);
			neurons[k].u += 1000.0 * dt * neurons[k].a *
								(0.2 * neurons[k].v - neurons[k].u);
			invals[k] = 0.0;
		}
		t_finish = clock();
		updatingneurons[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;

		/* check if spiked and calculate output */
		t_start = clock();
		for (k=0; k<n; k++) {
			outvals[k] = 0.0;
			if (neurons[k].v >= 30.0) {
				sr_save_spike(sr, k, t);
				outvals[k] = 1.0;
				neurons[k].v = -65.0;
				neurons[k].u += neurons[k].d;
				numspikes += 1;
			}
		}
		t_finish = clock();
		spikechecking[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;

		/* update neuron trace */		
		t_start = clock();
		for (k=0; k<n; k++) {
			if (k < n_exc) {
				spike_post[k] = outvals[k];
				trace_post[k] = trace_post[k]*(1.0 - (dt/tau_post)) +
								spike_post[k];
			}
		}
		t_finish = clock();
		updatingneutraces[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* update synapse strengths */
		t_start = clock();
		for (k=0; k<n; k++) {
			if (k < n_exc) {
				for (j=0; j < nums_in[k]; j++) {
					synapses[offsets[k]+j] = synapses[offsets[k]+j] + synbump +
							dt * (a_post * trace_pre[offsets[k]+j] * spike_post[k] -
								  a_pre * trace_post[k] * spike_pre[offsets[k]+j]);
					synapses[offsets[k]+j] =
						synapses[offsets[k]+j] < 0.0 ? 0.0 : synapses[offsets[k]+j];
					synapses[offsets[k]+j] =
						synapses[offsets[k]+j] > synmax ? synmax : synapses[offsets[k]+j];
					//spike_post[k] = 0.0;
				}
			}
		}
		t_finish = clock();
		updatingsynstrengths[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* push the output into the buffer */
		t_start = clock();
		for (k=0; k<n; k++)
			dn_pushoutput(outvals[k], k, dn);
		t_finish = clock();
		pushingoutput[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;


		/* advance the buffer */
		t_start = clock();
		dn_advance(dn);
		t_finish = clock();
		advancingbuffer[i] = ((double)(t_finish - t_start))/CLOCKS_PER_SEC;
	}

	/* performance analysis */
	double cycletime, cumtime = 0.0;

	printf("----------------------------------------\n");

	cycletime = 1000.0*dd_sum_double(gettinginputs, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Getting inputs:\t\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(updatingsyntraces, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Update syntraces:\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(updatingneurons, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Update neurons:\t\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(spikechecking, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Check spiked:\t\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(updatingneutraces, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Update neurtrace:\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(updatingsynstrengths, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Update syn strength:\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(pushingoutput, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Pushing buffer:\t\t %f (ms)\n", cycletime);

	cycletime = 1000.0*dd_sum_double(advancingbuffer, numsteps)/numsteps;
	cumtime += cycletime;
	printf("Advancing buffer:\t %f (ms)\n", cycletime);

	printf("Total cycle time:\t %f (ms)\n", cumtime);
	printf("Time per second: \t %f (ms)\n", cumtime*fs);

	/* Save spikes */
	FILE *spike_file;
	spike_file = fopen( "delnetstdp.dat", "w" );
	spike *firings = sr_spike_summary(sr);
	for (i=0; i<numspikes; i++)
		fprintf(spike_file, "%f  %d\n", firings[i].time, firings[i].neuron);
	fclose(spike_file);


	/* Clean up */
	dn_freedelnet(dn);
	sr_free(sr);
	free(g);
	free(firings);
	free(trace_post);
	free(spike_post);
	free(neurons);
	free(trace_pre);
	free(spike_pre);
	free(synapses);
	free(gettinginputs);
	free(updatingsyntraces);
	free(updatingneurons);
	free(spikechecking);
	free(updatingneutraces);
	free(updatingsynstrengths);
	free(pushingoutput);
	free(advancingbuffer);

	return 0;
}
